#include <stdio.h>
#include <math.h>
#include <malloc.h>
#include "hip/hip_runtime.h"

//#define M 12

double* polyfit(double* x, double* y, int n, int M)
{
	int m;
	m = n + 1;
	double **a = (double **)malloc(sizeof(double*)*m);
	for (int i = 0; i < m; i++)
	{
		a[i] = (double*)malloc(m * sizeof(double));
	}
	double *p = (double*)malloc(m * sizeof(double));
	double *b = (double*)malloc(m * sizeof(double));
	double *atemp = (double*)malloc(2 * m * sizeof(double));
	for (int i = 0; i < m; i++)
	{
		b[i] = 0;
		atemp[2 * i] = 0;
		atemp[2 * i + 1] = 0;
	}

	//	�������Է�����ϵ������b[]����

	for (int i = 0; i < M; i++)
	{
		for (int k = 1; k <= n * 2; k++)
		{
			atemp[k] += pow(x[i], k);
		}
		for (int h = 0; h < n + 1; h++)
		{
			b[h] += pow(x[i], h)*y[i];
		}
	}
	atemp[0] = M;

	for (int i = 0; i < m; i++)
	{
		int k = i;
		for (int g = 0; g < m; g++)
		{
			a[i][g] = atemp[k++];
		}
	}

	//�ⷽ�� a*p = b��
	//��������	
	for (int i = 0; i < n; i++)
	{
		if (a[i][i] == 0)
		{
			double temp = a[i][i];
			int idx = i;
			while (temp == 0)
			{
				temp = a[idx + 1][i];
				idx = idx + 1;
			}
			//������i�к͵�idx��
			double change, change_b;
			for (int g = 0; g < m; g++)
			{
				change = a[i][g];
				a[i][g] = a[idx][g];
				a[idx][g] = change;
			}
			change_b = b[i];
			b[i] = b[idx];
			b[idx] = change_b;
		}

		if (a[i][i] != 1)
		{
			double temp = a[i][i];
			for (int g = 0; g < m; g++)
			{
				a[i][g] = a[i][g] / temp;
			}
			b[i] = b[i] / temp;
		}

		for (int k = i + 1; k < m; k++)
		{
			if (a[k][i] != 0)
			{
				double temp = -a[k][i];
				for (int p = 0; p < m; p++)
				{
					a[k][p] = a[k][p] + a[i][p] * temp;
				}
				b[k] = b[k] + b[i] * temp;
			}
		}
	}

	if (a[n][n] != 1)
	{
		double temp = a[n][n];
		a[n][n] = 1;
		b[n] = b[n] / temp;
	}

	//��Ԫ
	for (int i = n; i > 0; i--)
	{
		for (int g = 0; g < i; g++)
		{
			double temp = -a[g][i];
			b[g] = b[g] + b[i] * temp;
		}
	}

	for (int i = 0; i < m; i++)
	{
		p[i] = b[n - i];
	}

	free(b);
	free(atemp);
	for (int i = 0; i < m; i++)
		free(a[i]);/*�ͷ���*/

	free(a);/*�ͷ���*/
	return p;
	free(p);

}