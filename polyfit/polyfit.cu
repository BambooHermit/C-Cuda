#include <stdio.h>
#include <math.h>
#include <malloc.h>
#include "hip/hip_runtime.h"

//#define M 12

double* polyfit(double* x, double* y, int n, int M)
{
	int m;
	m = n + 1;
	double **a = (double **)malloc(sizeof(double*)*m);
	for (int i = 0; i < m; i++)
	{
		a[i] = (double*)malloc(m * sizeof(double));
	}
	double *p = (double*)malloc(m * sizeof(double));
	double *b = (double*)malloc(m * sizeof(double));
	double *atemp = (double*)malloc(2 * m * sizeof(double));
	for (int i = 0; i < m; i++)
	{
		b[i] = 0;
		atemp[2 * i] = 0;
		atemp[2 * i + 1] = 0;
	}

	//	构建线性方程组系数矩阵，b[]不变

	for (int i = 0; i < M; i++)
	{
		for (int k = 1; k <= n * 2; k++)
		{
			atemp[k] += pow(x[i], k);
		}
		for (int h = 0; h < n + 1; h++)
		{
			b[h] += pow(x[i], h)*y[i];
		}
	}
	atemp[0] = M;

	for (int i = 0; i < m; i++)
	{
		int k = i;
		for (int g = 0; g < m; g++)
		{
			a[i][g] = atemp[k++];
		}
	}

	//解方程 a*p = b；
	//变上三角	
	for (int i = 0; i < n; i++)
	{
		if (a[i][i] == 0)
		{
			double temp = a[i][i];
			int idx = i;
			while (temp == 0)
			{
				temp = a[idx + 1][i];
				idx = idx + 1;
			}
			//交换第i行和第idx行
			double change, change_b;
			for (int g = 0; g < m; g++)
			{
				change = a[i][g];
				a[i][g] = a[idx][g];
				a[idx][g] = change;
			}
			change_b = b[i];
			b[i] = b[idx];
			b[idx] = change_b;
		}

		if (a[i][i] != 1)
		{
			double temp = a[i][i];
			for (int g = 0; g < m; g++)
			{
				a[i][g] = a[i][g] / temp;
			}
			b[i] = b[i] / temp;
		}

		for (int k = i + 1; k < m; k++)
		{
			if (a[k][i] != 0)
			{
				double temp = -a[k][i];
				for (int p = 0; p < m; p++)
				{
					a[k][p] = a[k][p] + a[i][p] * temp;
				}
				b[k] = b[k] + b[i] * temp;
			}
		}
	}

	if (a[n][n] != 1)
	{
		double temp = a[n][n];
		a[n][n] = 1;
		b[n] = b[n] / temp;
	}

	//消元
	for (int i = n; i > 0; i--)
	{
		for (int g = 0; g < i; g++)
		{
			double temp = -a[g][i];
			b[g] = b[g] + b[i] * temp;
		}
	}

	for (int i = 0; i < m; i++)
	{
		p[i] = b[n - i];
	}

	free(b);
	free(atemp);
	for (int i = 0; i < m; i++)
		free(a[i]);/*释放列*/

	free(a);/*释放行*/
	return p;
	free(p);

}