#include "hip/hip_runtime.h"

#include "hipfft/hipfft.h"
#include <stdio.h>
#include <malloc.h>
#include <math.h>
#include <complex>

#define BATCH 1

void fft(hipfftDoubleComplex *in, hipfftDoubleComplex *out, int size)
{

	hipfftDoubleComplex *inDev;
	hipfftDoubleComplex *outDev;
	hipMalloc((void **)&inDev, sizeof(hipfftDoubleComplex)*size);
	hipMalloc((void **)&outDev, sizeof(hipfftDoubleComplex)*size);
	hipfftHandle plan;
	hipfftPlan1d(&plan, size, HIPFFT_Z2Z, BATCH);

	hipMemcpy(inDev, in, sizeof(hipfftDoubleComplex)*size, hipMemcpyHostToDevice);
	hipfftExecZ2Z(plan, inDev, outDev, -1);
	hipDeviceSynchronize();
	hipMemcpy(out, outDev, sizeof(hipfftDoubleComplex)*size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	hipfftDestroy(plan);
	hipFree(inDev);
	hipFree(outDev);
}


void ifft(hipfftDoubleComplex *in, hipfftDoubleComplex *out, int size)
{

	hipfftDoubleComplex *inDev;
	hipfftDoubleComplex *outDev;
	hipMalloc((void **)&inDev, sizeof(hipfftDoubleComplex)*size);
	hipMalloc((void **)&outDev, sizeof(hipfftDoubleComplex)*size);
	hipfftDoubleComplex N;
	N.x = (double)size;
	N.y = (double)size;
	hipfftHandle plan;
	hipfftPlan1d(&plan, size, HIPFFT_Z2Z, BATCH);

	hipMemcpy(inDev, in, sizeof(hipfftDoubleComplex)*size, hipMemcpyHostToDevice);
	hipfftExecZ2Z(plan, inDev, outDev, 1);
	hipDeviceSynchronize();
	hipMemcpy(out, outDev, sizeof(hipfftDoubleComplex)*size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	for (int i = 0; i < size; i++)
	{
		out[i].x = out[i].x / N.x;
		out[i].y = out[i].y / N.y;
	}

	hipfftDestroy(plan);
	hipFree(inDev);
	hipFree(outDev);
}

void fftshift(hipfftDoubleComplex *in, hipfftDoubleComplex *out, int size)
{
	int idx = size / 2;

	for (int i = 0; i < idx; i++)
	{
		out[i] = in[size - idx + i];
		out[idx + i] = in[i];
	}

	if ((size % 2) == 1)
	{
		out[size - 1] = in[idx];
	}
}

void ifftshift(hipfftDoubleComplex *in, hipfftDoubleComplex *out, int size)
{
	int idx = size / 2;

	for (int i = 0; i < idx; i++)
	{
		out[size - idx + i] = in[i];
		out[i] = in[idx + i];
	}

	if ((size % 2) == 1)
	{
		out[idx] = in[size - 1];
	}

}